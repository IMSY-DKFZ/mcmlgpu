/////////////////////////////////////////////////////////////////////////////////////////////////////////
//
//   GPU-based Monte Carlo simulation of photon migration in multi-layered media (GPU-MCML)
//   Copyright (C) 2009
//	
//   || DEVELOPMENT TEAM: 
//   --------------------------------------------------------------------------------------------------
//   Erik Alerstam, David Han, and William C. Y. Lo
//   
//   This code is the result of the collaborative efforts between 
//   Lund University and the University of Toronto.  
//
//   || DOCUMENTATION AND USER MANUAL: 
//   --------------------------------------------------------------------------------------------------
//	 Detailed "Wiki" style documentation is being developed for GPU-MCML 
//   and will be available on our webpage soon:
//   http://code.google.com/p/gpumcml 
// 
//   || NEW FEATURES: 
//   --------------------------------------------------------------------------------------------------
//    - Supports the Fermi GPU architecture 
//    - Multi-GPU execution 
//    - Automatic selection of optimization parameters  
//    - Backward compatible on pre-Fermi graphics cards
//    - Supports linux and Windows environment (Visual Studio)
//   
//   || PREVIOUS WORK: 
//   --------------------------------------------------------------------------------------------------
//	 This code is the fusion of our earlier, preliminary implementations and combines the best features 
//   from each implementation.  
//
//   W. C. Y. Lo, T. D. Han, J. Rose, and L. Lilge, "GPU-accelerated Monte Carlo simulation for photodynamic
//   therapy treatment planning," in Proc. of SPIE-OSA Biomedical Optics, vol. 7373.
//   
//   and 
//
//   http://www.atomic.physics.lu.se/biophotonics/our_research/monte_carlo_simulations/gpu_monte_carlo/
//	 E. Alerstam, T. Svensson and S. Andersson-Engels, "Parallel computing with graphics processing
//	 units for high-speed Monte Carlo simulations of photon migration", Journal of Biomedical Optics
//	 Letters, 13(6) 060504 (2008).
//
//   || CITATION: 
//   --------------------------------------------------------------------------------------------------
//	 We encourage the use, and modification of this code, and hope it will help 
//	 users/programmers to utilize the power of GPGPU for their simulation needs. While we
//	 don't have a scientific publication describing this code yet, we would very much appreciate it
//	 if you cite our original papers above if you use this code or derivations 
//   thereof for your own scientific work
//
//	 To compile and run this code, please visit www.nvidia.com and download the necessary 
//	 CUDA Toolkit, SDK, and Developer Drivers 
//
//	 If you use Visual Studio, the express edition is available for free at 
//   http://www.microsoft.com/express/Downloads/). 
//  	
//   This code is distributed under the terms of the GNU General Public Licence (see below). 
//
/////////////////////////////////////////////////////////////////////////////////////////////////////////
/*	 
*   This file is part of GPUMCML.
* 
*   GPUMCML is free software: you can redistribute it and/or modify
*   it under the terms of the GNU General Public License as published by
*   the Free Software Foundation, either version 3 of the License, or
*   (at your option) any later version.
*
*   GPUMCML is distributed in the hope that it will be useful,
*   but WITHOUT ANY WARRANTY; without even the implied warranty of
*   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
*   GNU General Public License for more details.
*
*   You should have received a copy of the GNU General Public License
*   along with GPUMCML.  If not, see <http://www.gnu.org/licenses/>.
*/


#include <float.h> //for FLT_MAX 
#include <stdio.h>
#include <string.h>

#include <hip/hip_runtime.h>


#ifdef _WIN32
#include "gpumcml_io.c"
#include "cutil.h"
#else 
#include <cutil.h>
#endif

#ifdef _WIN32 
#include "cutil-win32/multithreading.h"
#else
#include "multithreading.h"
#endif

#include "gpumcml.h"
#include "gpumcml_kernel.h"

#include "gpumcml_kernel.cu"
#include "gpumcml_mem.cu"
//////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////
//   Supports multiple GPUs by allowing multiple host threads to launch kernel
//   Each thread calls RunGPUi with its own HostThreadState parameters
//////////////////////////////////////////////////////////////////////////////
static CUT_THREADPROC RunGPUi(HostThreadState *hstate)
{
  SimState *HostMem = &(hstate->host_sim_state);
  SimState DeviceMem;
  GPUThreadStates tstates;
  // total number of threads in the grid
  UINT32 n_threads = hstate->n_tblks * NUM_THREADS_PER_BLOCK;
  hipError_t cudastat;

  CUDA_SAFE_CALL( hipSetDevice(hstate->dev_id) );

  // Init the remaining states.
  InitSimStates(HostMem, &DeviceMem, &tstates, hstate->sim, n_threads);
  CUDA_SAFE_CALL( hipDeviceSynchronize() ); // Wait for all threads to finish
  cudastat=hipGetLastError(); // Check if there was an error
  if (cudastat)
  {
    fprintf(stderr, "[GPU %u] failure in InitSimStates (%i): %s\n",
      hstate->dev_id, cudastat, hipGetErrorString(cudastat));
    FreeHostSimState(HostMem);
    FreeDeviceSimStates(&DeviceMem, &tstates);
    exit(1); 
  }

  InitDCMem(hstate->sim, hstate->A_rz_overflow);
  CUDA_SAFE_CALL( hipDeviceSynchronize() ); // Wait for all threads to finish
  cudastat=hipGetLastError(); // Check if there was an error
  if (cudastat)
  {
    fprintf(stderr, "[GPU %u] failure in InitDCMem (%i): %s\n",
      hstate->dev_id, cudastat, hipGetErrorString(cudastat));
    FreeHostSimState(HostMem);
    FreeDeviceSimStates(&DeviceMem, &tstates);
    exit(1); 
  }

  dim3 dimBlock(NUM_THREADS_PER_BLOCK);
  dim3 dimGrid(hstate->n_tblks);

  // Initialize the remaining thread states.
  InitThreadState<<<dimGrid,dimBlock>>>(tstates, *HostMem->n_photons_left);
  CUDA_SAFE_CALL( hipDeviceSynchronize() ); // Wait for all threads to finish
  cudastat=hipGetLastError(); // Check if there was an error
  if (cudastat)
  {
    fprintf(stderr, "[GPU %u] failure in InitThreadState (%i): %s\n",
      hstate->dev_id, cudastat, hipGetErrorString(cudastat));
    FreeHostSimState(HostMem);
    FreeDeviceSimStates(&DeviceMem, &tstates);
    exit(1); 
  }

#if !defined(CACHE_A_RZ_IN_SMEM) && __CUDA_ARCH__ >= 200
  // Configure the L1 cache for Fermi.
  if (hstate->sim->ignoreAdetection == 1)
  {
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(MCMLKernel<1>), hipFuncCachePreferL1);
  }
  else
  {
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(MCMLKernel<0>), hipFuncCachePreferL1);
  }
#endif

  int k_smem_sz = 0;
#ifdef USE_32B_ELEM_FOR_ARZ_SMEM
  // This piece of shared memory is for overflow handling.
  k_smem_sz = NUM_THREADS_PER_BLOCK * sizeof(UINT32);
#endif

  for (int i = 1; *HostMem->n_photons_left > 0; ++i)
  {
    // Run the kernel.
    if (hstate->sim->ignoreAdetection == 1)
    {
      MCMLKernel<1><<<dimGrid, dimBlock, k_smem_sz>>>(DeviceMem, tstates);
    }
    else
    {
      MCMLKernel<0><<<dimGrid, dimBlock, k_smem_sz>>>(DeviceMem, tstates);
    }
    // Wait for all threads to finish.
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    // Check if there was an error
    cudastat = hipGetLastError();
    if (cudastat)
    {
      fprintf(stderr, "[GPU %u] failure in MCMLKernel (%i): %s.\n",
        hstate->dev_id, cudastat, hipGetErrorString(cudastat));
      FreeHostSimState(HostMem);
      FreeDeviceSimStates(&DeviceMem, &tstates);
      exit(1); 
    }

    // Copy the number of photons left from device to host.
    CUDA_SAFE_CALL( hipMemcpy(HostMem->n_photons_left,
      DeviceMem.n_photons_left, sizeof(unsigned int),
      hipMemcpyDeviceToHost) );

    // printf("[GPU %u] batch %5d, number of photons left %10u\n",
      // hstate->dev_id, i, *(HostMem->n_photons_left));
  }

  // Sum the multiple copies of A_rz in the global memory.
  sum_A_rz<<<30, 128>>>(DeviceMem.A_rz);
  // Wait for all threads to finish.
  CUDA_SAFE_CALL( hipDeviceSynchronize() );
  // Check if there was an error
  cudastat = hipGetLastError();
  if (cudastat)
  {
    fprintf(stderr, "[GPU %u] failure in sum_A_rz (%i): %s.\n",
        hstate->dev_id, cudastat, hipGetErrorString(cudastat));
    FreeHostSimState(HostMem);
    FreeDeviceSimStates(&DeviceMem, &tstates);
    exit(1); 
  }

  // printf("[GPU %u] simulation done!\n", hstate->dev_id);

  CopyDeviceToHostMem(HostMem, &DeviceMem, hstate->sim, n_threads);
  FreeDeviceSimStates(&DeviceMem, &tstates);
  // We still need the host-side structure.
}

//////////////////////////////////////////////////////////////////////////////
//   Perform MCML simulation for one run out of N runs (in the input file)
//////////////////////////////////////////////////////////////////////////////
static double DoOneSimulation(int sim_id, SimulationStruct* simulation,
                            HostThreadState* hstates[], UINT32 num_GPUs,
                            UINT64 *x, UINT32 *a, char* mcoFile, SimulationResults* simResults)
{
  // printf("\n------------------------------------------------------------\n");
  // printf("        Simulation #%d\n", sim_id);
  // printf("        - number_of_photons = %u\n", simulation->number_of_photons);

  // Compute GPU-specific constant parameters.
  UINT32 A_rz_overflow = 0;
  double elapsedTime = 0.;
  // We only need it if we care about A_rz.
#if defined(CACHE_A_RZ_IN_SMEM) && defined(USE_32B_ELEM_FOR_ARZ_SMEM)
  if (! simulation->ignoreAdetection)
  {
    A_rz_overflow = compute_Arz_overflow_count(simulation->start_weight,
        simulation->layers, simulation->n_layers, NUM_THREADS_PER_BLOCK);
    printf("        - A_rz_overflow = %u\n", A_rz_overflow);
  }
#endif

  // printf("------------------------------------------------------------\n\n");

  // Start simulation kernel exec timer
  unsigned int execTimer = 0;
  CUT_SAFE_CALL( cutCreateTimer(&execTimer) );
  CUT_SAFE_CALL( cutStartTimer(execTimer) );

  // Distribute all photons among GPUs.
  UINT32 n_photons_per_GPU = simulation->number_of_photons / num_GPUs;

  // For each GPU, init the host-side structure.
  for (UINT32 i = 0; i < num_GPUs; ++i)
  {
    hstates[i]->sim = simulation;
    hstates[i]->A_rz_overflow = A_rz_overflow;

    SimState *hss = &(hstates[i]->host_sim_state);

    // number of photons responsible 
    hss->n_photons_left = (UINT32*)malloc(sizeof(UINT32));
    // The last GPU may be responsible for more photons if the
    // distribution is uneven.
    *(hss->n_photons_left) = (i == num_GPUs-1) ?
      simulation->number_of_photons - (num_GPUs-1) * n_photons_per_GPU :
      n_photons_per_GPU;
  }

  // Launch a dedicated host thread for each GPU.
  CUTThread hthreads[MAX_GPU_COUNT];
  for (UINT32 i = 0; i < num_GPUs; ++i)
  {
    hthreads[i] = cutStartThread((CUT_THREADROUTINE)RunGPUi, hstates[i]);
  }

  // Wait for all host threads to finish.
  cutWaitForThreads(hthreads, num_GPUs);


  // Check any of the threads failed.
  int failed = 0;
  for (UINT32 i = 0; i < num_GPUs && !failed; ++i)
  {
    if (hstates[i]->host_sim_state.n_photons_left == NULL) failed = 1;
  }

  if (!failed)
  {
    // Sum the results to hstates[0].
    SimState *hss0 = &(hstates[0]->host_sim_state);
    for (UINT32 i = 1; i < num_GPUs; ++i)
    {
      SimState *hssi = &(hstates[i]->host_sim_state);

      // A_rz
      int size = simulation->det.nr * simulation->det.nz;
      for (int j = 0; j < size; ++j)
      {
        hss0->A_rz[j] += hssi->A_rz[j];
      }

      // Rd_ra
      size = simulation->det.na * simulation->det.nr;
      for (int j = 0; j < size; ++j)
      {
        hss0->Rd_ra[j] += hssi->Rd_ra[j];
      }

      // Tt_ra
      size = simulation->det.na * simulation->det.nr;
      for (int j = 0; j < size; ++j)
      {
        hss0->Tt_ra[j] += hssi->Tt_ra[j];
      }
    }

    CUT_SAFE_CALL( cutStopTimer(execTimer) );

    elapsedTime = static_cast<double>(cutGetTimerValue(execTimer));
    // printf("\n\n>>>>>>Simulation time: %.3f ms\n", elapsedTime);
    // TODO: create class to store results and write them only at the end of all simulations
    // could create a method within a class called registerSimulationResult
    //Write_Simulation_Results(hss0, simulation, elapsedTime, mcoFile);
    simResults->registerSimulationResults(hss0, simulation)
  }

  CUT_SAFE_CALL( cutDeleteTimer(execTimer) );

  // Free SimState structs.
  for (UINT32 i = 0; i < num_GPUs; ++i)
  {
    FreeHostSimState(&(hstates[i]->host_sim_state));
  }
  return elapsedTime;
}


//////////////////////////////////////////////////////////////////////////////
//   Perform MCML simulation for one run out of N runs (in the input file)
//////////////////////////////////////////////////////////////////////////////
int main(int argc, char* argv[])
{
  char* filename = NULL;
  char* mcoFile = NULL;
  UINT64 seed = (UINT64) time(NULL);
  int ignoreAdetection = 0;
  char* mcoFolder = NULL;
  UINT32 num_GPUs = 1;
  FILE* pFile_outp;

  SimulationStruct* simulations;
  int n_simulations;
  int i;
  double elapsedTime = 0.;

  // Parse command-line arguments.
  if (interpret_arg(argc, argv, &filename,
    &seed, &ignoreAdetection, &num_GPUs, &mcoFolder))
  {
    usage(argv[0]);
    return 1;
  }

  // Determine the number of GPUs available.
  int dev_count;
  CUDA_SAFE_CALL( hipGetDeviceCount(&dev_count) );
  if (dev_count <= 0)
  {
    fprintf(stderr, "No GPU available. Quit.\n");
    return 1;
  }

  // Make sure we do not use more than what we have.
  if (num_GPUs > dev_count)
  {
    printf("The number of GPUs specified (%u) is more than "
      "what is available (%d)!\n", num_GPUs, dev_count);
    num_GPUs = (UINT32)dev_count;
  }

  // Output the execution configuration.
  printf("\n====================================\n");
  printf("EXECUTION MODE:\n");
  printf("  ignore A-detection:      %s\n",
    ignoreAdetection ? "YES" : "NO");
  printf("  seed:                    %llu\n", seed);
  printf("  # of GPUs:               %u\n", num_GPUs);
  printf("====================================\n\n");

  // Read the simulation inputs.
  n_simulations = read_simulation_data(filename, &simulations,
    ignoreAdetection);
  if(n_simulations == 0)
  {
    printf("Something wrong with read_simulation_data!\n");
    return 1;
  }
  printf("Read %d simulations\n\n",n_simulations);

  // Allocate one host thread state for each GPU.
  HostThreadState* hstates[MAX_GPU_COUNT];
  hipDeviceProp_t props;
  int n_threads = 0;    // total number of threads for all GPUs
  for (i = 0; i < num_GPUs; ++i)
  {
    hstates[i] = (HostThreadState*)malloc(sizeof(HostThreadState));

    // Set the GPU ID.
    hstates[i]->dev_id = i;

    // Get the GPU properties.
    CUDA_SAFE_CALL( hipGetDeviceProperties(&props, hstates[i]->dev_id) );
    printf("[GPU %u] \"%s\" with Compute Capability %d.%d (%d SMs)\n",
        i, props.name, props.major, props.minor, props.multiProcessorCount);

    // Validate the GPU compute capability.
    int cc = (props.major * 10 + props.minor) * 10;
    if (cc < __CUDA_ARCH__)
    {
      fprintf(stderr, "\nGPU %u does not meet the Compute Capability "
          "this program requires (%d)! Abort.\n\n", i, __CUDA_ARCH__);
      exit(1);
    }

    // We launch one thread block for each SM on this GPU.
    hstates[i]->n_tblks = props.multiProcessorCount;

    n_threads += hstates[i]->n_tblks * NUM_THREADS_PER_BLOCK;
  }

  // Allocate and initialize RNG seeds (for all threads on all GPUs).
  UINT64 *x = (UINT64*)malloc(n_threads * sizeof(UINT64));
  UINT32 *a = (UINT32*)malloc(n_threads * sizeof(UINT32));

#ifdef _WIN32 
    if (init_RNG(x, a, n_threads, "safeprimes_base32.txt", seed)) return 1;
#else 
    if (init_RNG(x, a, n_threads, "executable/safeprimes_base32.txt", seed)) return 1;
#endif

  printf("\nUsing the MWC random number generator ...\n");

  // Assign these seeds to each host thread state.
  int ofst = 0;
  for (i = 0; i < num_GPUs; ++i)
  {
    SimState *hss = &(hstates[i]->host_sim_state);
    hss->x = &x[ofst];
    hss->a = &a[ofst];

    ofst += hstates[i]->n_tblks * NUM_THREADS_PER_BLOCK;
  }

  // write file header
  mcoFile = strcat(mcoFolder, "/batch.mco");
  pFile_outp = fopen (mcoFile , "w");
  fprintf(pFile_outp, "ID,Specular,Diffuse,Absorbed,Transmittance,Penetration\n");
  fclose(pFile_outp);

  SimulationResults simResults;
  //perform all the simulations
  for(int i = 0; i < n_simulations; i++)
  {
    // Run a simulation
    elapsedTime = DoOneSimulation(i, &simulations[i], hstates, num_GPUs, x, a, mcoFile, &simResults);
    printProgress(i / float(n_simulations), elapsedTime);
  }
  simResults.writeSimulationResults(mcoFile)
  // Free host thread states.
  for (i = 0; i < num_GPUs; ++i) free(hstates[i]);

  // Free the random number seed arrays.
  free(x); free(a);

  FreeSimulationStruct(simulations, n_simulations);

  return 0; 
}

//////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////

