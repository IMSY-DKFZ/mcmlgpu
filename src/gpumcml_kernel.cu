#include "hip/hip_runtime.h"
/*****************************************************************************
*
*   Kernel code for GPUMCML
*   =========================================================================
*   Featured Optimizations: 
*   1) Shared memory cache for high fluence region  
*   2) Reduced divergence 
*   3) Optimized atomicAdd
*
****************************************************************************/
/*	 
*   This file is part of GPUMCML.
* 
*   GPUMCML is free software: you can redistribute it and/or modify
*   it under the terms of the GNU General Public License as published by
*   the Free Software Foundation, either version 3 of the License, or
*   (at your option) any later version.
*
*   GPUMCML is distributed in the hope that it will be useful,
*   but WITHOUT ANY WARRANTY; without even the implied warranty of
*   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
*   GNU General Public License for more details.
*
*   You should have received a copy of the GNU General Public License
*   along with GPUMCML.  If not, see <http://www.gnu.org/licenses/>.
*/

#ifndef _GPUMCML_KERNEL_CU_
#define _GPUMCML_KERNEL_CU_

#include "gpumcml_kernel.h"
#include "gpumcml_rng.cu"

// We use different math intrinsics for single- and double-precision.
#ifdef SINGLE_PRECISION
#define FAST_DIV(x,y) __fdividef(x,y)
#define SQRT(x) sqrtf(x)
#define RSQRT(x) rsqrtf(x)
#define LOG(x) logf(x)
#define SINCOS(x, sptr, cptr) __sincosf(x, sptr, cptr)
#else
// __CUDA_ARCH__ is defined by the CUDA compiler.
#if __CUDA_ARCH__ >= 200
#define FAST_DIV(x,y) __ddiv_rn(x,y)
#else
#define FAST_DIV(x,y) (x/y)
#endif
#define SQRT(x) sqrt(x)
#define RSQRT(x) rsqrt(x)
#define LOG(x) log(x)
#define SINCOS(x, sptr, cptr) sincos(x, sptr, cptr)
#endif

//////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////
// This host routine computes the maximum element value of A_rz in shared
// memory, that indicates an imminent overflow.
//
// This MAX_OVERFLOW is MAX_UINT32 - MAX(dwa) * NUM_THREADS_PER_BLOCK.
//
// All we really need to compute is
//    MAX(dwa) <= WEIGHT_SCALE * <init_photon_w> * MAX( mua/(mua+mus) )
//
// We have to be accurate in this bound because if we assume that
//    MAX(dwa) = WEIGHT_SCALE,
// MAX_OVERFLOW can be small if WEIGHT_SCALE is large, like 12000000.
//
// <n_layers> is the length of <layers>, excluding the top and bottom layers.
//////////////////////////////////////////////////////////////////////////////
UINT32 compute_Arz_overflow_count(GFLOAT init_photon_w,
        LayerStruct *layers, UINT32 n_layers, UINT32 n_threads_per_tblk)
{
    // Determine the largest mua/(mua+mus) over all layers.
    double max_muas = 0;
    for (int i = 1; i <= n_layers; ++i)
    {
        double muas = layers[i].mua * layers[i].mutr;
        if (max_muas < muas) max_muas = muas;
    }

    // Determine an upper bound of <dwa> in <MCMLKernel>.
    UINT32 max_dwa = (UINT32)(init_photon_w * max_muas * WEIGHT_SCALE) + 1;

    return (0xFFFFFFFF - max_dwa * n_threads_per_tblk);
}

//////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////
//   Initialize photon position (x, y, z), direction (ux, uy, uz), weight (w), 
//   and current layer (layer) 
//   Note: Infinitely narrow beam (pointing in the +z direction = downwards)
//////////////////////////////////////////////////////////////////////////////
__device__ void LaunchPhoton(PhotonStructGPU *photon)
{
  photon->x = photon->y = photon->z = MCML_FP_ZERO;
  photon->ux = photon->uy = MCML_FP_ZERO;
  photon->uz = FP_ONE;
  photon->w = d_simparam.init_photon_w;
  photon->layer = 1;
}

//////////////////////////////////////////////////////////////////////////////
//   Initialize thread states (tstates), created to allow a large 
//   simulation to be broken up into batches 
//   (avoiding display driver time-out errors)
//////////////////////////////////////////////////////////////////////////////
__global__ void InitThreadState(GPUThreadStates tstates, UINT32 n_photons)
{
  PhotonStructGPU photon_temp; 

  // thread ID that is unique in the grid
  UINT32 tid = blockIdx.x * blockDim.x + threadIdx.x;

  // If the total number of threads exceeds the number of photons, some
  // threads will not do any work.
  int is_active = (tid < n_photons) ? 1 : 0;
  tstates.is_active[tid] = is_active;

  if (is_active)
  {
    // Initialize the photon and copy into photon_<parameter x>
    LaunchPhoton(&photon_temp);

    tstates.photon_x[tid] = photon_temp.x;
    tstates.photon_y[tid] = photon_temp.y;
    tstates.photon_z[tid] = photon_temp.z;
    tstates.photon_ux[tid] = photon_temp.ux;
    tstates.photon_uy[tid] = photon_temp.uy;
    tstates.photon_uz[tid] = photon_temp.uz;
    tstates.photon_w[tid] = photon_temp.w;
    tstates.photon_layer[tid] = photon_temp.layer;
  }
}

//////////////////////////////////////////////////////////////////////////////
//   Save thread states (tstates), by copying the current photon 
//   data from registers into global memory
//////////////////////////////////////////////////////////////////////////////
__device__ void SaveThreadState(SimState *d_state, GPUThreadStates *tstates,
                                PhotonStructGPU *photon,
                                UINT64 rnd_x, 
                                UINT32 is_active)
{
  UINT32 tid = blockIdx.x * blockDim.x + threadIdx.x;

  d_state->x[tid] = rnd_x;

  tstates->photon_x[tid] = photon->x;
  tstates->photon_y[tid] = photon->y;
  tstates->photon_z[tid] = photon->z;
  tstates->photon_ux[tid] = photon->ux;
  tstates->photon_uy[tid] = photon->uy;
  tstates->photon_uz[tid] = photon->uz;
  tstates->photon_w[tid] = photon->w;
  tstates->photon_layer[tid] = photon->layer;

  tstates->is_active[tid] = is_active;
}

//////////////////////////////////////////////////////////////////////////////
//   Restore thread states (tstates), by copying the latest photon 
//   data from global memory back into the registers
//////////////////////////////////////////////////////////////////////////////
__device__ void RestoreThreadState(SimState *d_state, GPUThreadStates *tstates,
                                   PhotonStructGPU *photon,
                                   UINT64 *rnd_x, UINT32 *rnd_a,
                                   UINT32 *is_active)
{
  UINT32 tid = blockIdx.x * blockDim.x + threadIdx.x;

  *rnd_x = d_state->x[tid];
  *rnd_a = d_state->a[tid]; 

  photon->x = tstates->photon_x[tid];
  photon->y = tstates->photon_y[tid];
  photon->z = tstates->photon_z[tid];
  photon->ux = tstates->photon_ux[tid];
  photon->uy = tstates->photon_uy[tid];
  photon->uz = tstates->photon_uz[tid];
  photon->w = tstates->photon_w[tid];
  photon->layer = tstates->photon_layer[tid];

  *is_active = tstates->is_active[tid];
}

//////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////

#ifdef CACHE_A_RZ_IN_SMEM

//////////////////////////////////////////////////////////////////////////////
// Flush the element at offset <s_addr> of A_rz in shared memory (s_A_rz)
// to the global memory (g_A_rz). <s_A_rz> is of dimension MAX_IR x MAX_IZ.
//////////////////////////////////////////////////////////////////////////////
__device__ void Flush_Arz(UINT64 *g_A_rz, ARZ_SMEM_TY *s_A_rz, UINT32 saddr)
{
  UINT32 ir = saddr / MAX_IZ;
  UINT32 iz = saddr - ir * MAX_IZ;
  UINT32 g_addr = ir * d_simparam.nz + iz;

  atomicAdd(&g_A_rz[g_addr], (UINT64)s_A_rz[saddr]);
}

//////////////////////////////////////////////////////////////////////////////
//   AtomicAdd to Shared Mem for Unsigned Long Long (ULL) data type
//   Note: Only Fermi architecture supports 64-bit atomicAdd to shared memory
//////////////////////////////////////////////////////////////////////////////
__device__ void AtomicAddULL_Shared(UINT64* address, UINT32 add)
{
#ifdef USE_64B_ATOMIC_SMEM
  // TODO: does this really work?
  atomicAdd(address, (UINT64)add);
#else
  if (atomicAdd((UINT32*)address,add) +add < add)
  {
    atomicAdd(((UINT32*)address)+1, 1U);
  }
#endif
}

#endif  // CACHE_A_RZ_IN_SMEM

//////////////////////////////////////////////////////////////////////////////
//   AtomicAdd to Global Mem for Unsigned Long Long (ULL) data type
//   Note: 64-bit atomicAdd to global memory is supported since
//   Compute Capability 1.2
//////////////////////////////////////////////////////////////////////////////
__device__ void AtomicAddULL_Global(UINT64* address, UINT32 add)
{
#ifdef USE_64B_ATOMIC_GMEM
  atomicAdd(address, (UINT64)add);
#else
  if (atomicAdd((UINT32*)address,add) +add < add)
  {
    atomicAdd(((UINT32*)address)+1, 1U);
  }
#endif
}

//////////////////////////////////////////////////////////////////////////////
//   Compute the step size for a photon packet when it is in tissue
//   Calculate new step size: -log(rnd)/(mua+mus).
//////////////////////////////////////////////////////////////////////////////
__device__ void ComputeStepSize(PhotonStructGPU *photon,
                                UINT64 *rnd_x, UINT32 *rnd_a)
{
  photon->s = -LOG(rand_MWC_oc(rnd_x,rnd_a))
    * d_layerspecs[photon->layer].rmuas;
}


//////////////////////////////////////////////////////////////////////////////
//   Check if the step size calculated above will cause the photon to hit the 
//   boundary between 2 layers.
//   Return 1 for a hit, 0 otherwise.
//   If the projected step hits the boundary, the photon steps to the boundary
//////////////////////////////////////////////////////////////////////////////
__device__ int HitBoundary(PhotonStructGPU *photon)
{
  /* step size to boundary. */
  GFLOAT dl_b; 

  /* Distance to the boundary. */
  GFLOAT z_bound = (photon->uz > MCML_FP_ZERO) ?
    d_layerspecs[photon->layer].z1 : d_layerspecs[photon->layer].z0;
  dl_b = FAST_DIV(z_bound - photon->z, photon->uz);     // dl_b > 0

  UINT32 hit_boundary = (photon->uz != MCML_FP_ZERO) && (photon->s > dl_b);
  if (hit_boundary)
  {
    // No need to multiply by (mua + mus), as it is later
    // divided by (mua + mus) anyways (in the original version).
    photon->s = dl_b;
  }

  return hit_boundary;
}

//////////////////////////////////////////////////////////////////////////////
//   Move the photon by step size (s) along direction (ux,uy,uz) 
//////////////////////////////////////////////////////////////////////////////
__device__ void Hop(PhotonStructGPU *photon)
{
  photon->x += photon->s * photon->ux;
  photon->y += photon->s * photon->uy;
  photon->z += photon->s * photon->uz;
}

//////////////////////////////////////////////////////////////////////////////
//   UltraFast version (featuring reduced divergence compared to CPU-MCML)
//   If a photon hits a boundary, determine whether the photon is transmitted
//   into the next layer or reflected back by computing the internal reflectance
//////////////////////////////////////////////////////////////////////////////
__device__ void FastReflectTransmit(PhotonStructGPU *photon,
                                    SimState *d_state_ptr,
                                    UINT64 *rnd_x, UINT32 *rnd_a)
{
  /* Collect all info that depend on the sign of "uz". */
  GFLOAT cos_crit;
  UINT32 new_layer;
  if (photon->uz > MCML_FP_ZERO)
  {
    cos_crit = d_layerspecs[photon->layer].cos_crit1;
    new_layer = photon->layer+1;
  }
  else
  {
    cos_crit = d_layerspecs[photon->layer].cos_crit0;
    new_layer = photon->layer-1;
  }

  // cosine of the incident angle (0 to 90 deg)
  GFLOAT ca1 = fabsf(photon->uz);

  // The default move is to reflect.
  photon->uz = -photon->uz;

  // Moving this check down to "RFresnel = MCML_FP_ZERO" slows down the
  // application, possibly because every thread is forced to do
  // too much.
  if (ca1 > cos_crit)
  {
    /* Compute the Fresnel reflectance. */

    // incident and transmit refractive index
    GFLOAT ni = d_layerspecs[photon->layer].n;
    GFLOAT nt = d_layerspecs[new_layer].n;
    GFLOAT ni_nt = FAST_DIV(ni, nt);   // reused later

    GFLOAT sa1 = SQRT(FP_ONE-ca1*ca1);
    if (ca1 > COSZERO) sa1 = MCML_FP_ZERO;
    GFLOAT sa2 = fminf(ni_nt * sa1, FP_ONE);
    GFLOAT uz1 = SQRT(FP_ONE-sa2*sa2);    // uz1 = ca2

    GFLOAT ca1ca2 = ca1 * uz1;
    GFLOAT sa1sa2 = sa1 * sa2;
    GFLOAT sa1ca2 = sa1 * uz1;
    GFLOAT ca1sa2 = ca1 * sa2;

    // normal incidence: [(1-ni_nt)/(1+ni_nt)]^2
    // We ensure that ca1ca2 = 1, sa1sa2 = 0, sa1ca2 = 1, ca1sa2 = ni_nt
    if (ca1 > COSZERO)
    {
      sa1ca2 = FP_ONE;
      ca1sa2 = ni_nt;
    }

    GFLOAT cam = ca1ca2 + sa1sa2; /* c- = cc + ss. */
    GFLOAT sap = sa1ca2 + ca1sa2; /* s+ = sc + cs. */
    GFLOAT sam = sa1ca2 - ca1sa2; /* s- = sc - cs. */

    GFLOAT rFresnel = FAST_DIV(sam, sap*cam);
    rFresnel *= rFresnel;
    rFresnel *= (ca1ca2*ca1ca2 + sa1sa2*sa1sa2);

    // In this case, we do not care if "uz1" is exactly 0.
    if (ca1 < COSNINETYDEG || sa2 == FP_ONE) rFresnel = FP_ONE;

    GFLOAT rand = rand_MWC_co(rnd_x, rnd_a);

    if (rFresnel < rand)
    {
      // The move is to transmit.
      photon->layer = new_layer;

      // Let's do these even if the photon is dead.
      photon->ux *= ni_nt;
      photon->uy *= ni_nt;
      photon->uz = -copysignf(uz1, photon->uz);

      if (photon->layer == 0 || photon->layer > d_simparam.num_layers)
      {
        // transmitted
        GFLOAT uz2 = photon->uz;
        UINT64 *ra_arr = d_state_ptr->Tt_ra;
        if (photon->layer == 0)
        {
          // diffuse reflectance
          uz2 = -uz2;
          ra_arr = d_state_ptr->Rd_ra;
        }

        UINT32 ia = acosf(uz2) * FP_TWO * RPI * d_simparam.na;
        UINT32 ir = FAST_DIV(SQRT(photon->x*photon->x+photon->y*photon->y), d_simparam.dr);
        if (ir >= d_simparam.nr) ir = d_simparam.nr - 1;

        AtomicAddULL_Global(&ra_arr[ia * d_simparam.nr + ir],
          (UINT32)(photon->w * WEIGHT_SCALE));
 
        // Kill the photon.
        photon->w = MCML_FP_ZERO;
      }
    }
  }
}

//////////////////////////////////////////////////////////////////////////////
//   Computing the scattering angle and new direction by 
//	 sampling the polar deflection angle theta and the
// 	 azimuthal angle psi.
//////////////////////////////////////////////////////////////////////////////
__device__ void Spin(GFLOAT g, PhotonStructGPU *photon,
                     UINT64 *rnd_x, UINT32 *rnd_a)
{
  GFLOAT cost, sint; // cosine and sine of the polar deflection angle theta
  GFLOAT cosp, sinp; // cosine and sine of the azimuthal angle psi
  GFLOAT psi;
  GFLOAT temp;
  GFLOAT last_ux, last_uy, last_uz;
  GFLOAT rand;

  /***********************************************************
  *	>>>>>>> SpinTheta
  *  Choose (sample) a new theta angle for photon propagation
  *	according to the anisotropy.
  *
  *	If anisotropy g is 0, then
  *		cos(theta) = 2*rand-1.
  *	otherwise
  *		sample according to the Henyey-Greenstein function.
  *
  *	Returns the cosine of the polar deflection angle theta.
  ****/

  rand = rand_MWC_oc(rnd_x, rnd_a); 

  cost = FP_TWO * rand - FP_ONE;

  if (g != MCML_FP_ZERO)
  {
    temp = FAST_DIV((FP_ONE - g * g), FP_ONE + g*cost);
    cost = FAST_DIV(FP_ONE + g * g - temp*temp, FP_TWO * g);
    //cost = fmaxf(cost, -FP_ONE); //these are just here because of the bad PRNG in MCML
    //cost = fminf(cost, FP_ONE);
  }
  sint = SQRT(FP_ONE - cost * cost);

  /* spin psi 0-2pi. */
  rand = rand_MWC_co(rnd_x, rnd_a);

  psi = FP_TWO * PI_const * rand;
  SINCOS(psi, &sinp, &cosp);

  GFLOAT stcp = sint * cosp;
  GFLOAT stsp = sint * sinp;

  last_ux = photon->ux;
  last_uy = photon->uy;
  last_uz = photon->uz;

  if (fabsf(last_uz) > COSZERO) 
  // Normal incident.
  {
    photon->ux = stcp;
    photon->uy = stsp;
    photon->uz = copysignf(cost,last_uz*cost); 
  }
  else 
  // Regular incident. 
  {
    temp = RSQRT(FP_ONE - last_uz * last_uz);
    photon->ux = (stcp * last_ux * last_uz - stsp * last_uy) * temp
      + last_ux * cost;
    photon->uy = (stcp * last_uy * last_uz + stsp * last_ux) * temp
      + last_uy * cost;
    photon->uz = FAST_DIV(-stcp, temp) + last_uz * cost;
  }

  // Normalize unit vector to ensure its magnitude is 1 (unity) 
  // only required in 32-bit floating point version
#ifdef SINGLE_PRECISION
  temp=RSQRT(photon->ux*photon->ux+photon->uy*photon->uy+photon->uz*photon->uz);
  photon->ux = photon->ux *temp;
  photon->uy = photon->uy *temp;
  photon->uz = photon->uz *temp;
#endif
}

//////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////

extern __shared__ UINT32 MCMLKernel_smem[];

//////////////////////////////////////////////////////////////////////////////
//   Main Kernel for MCML (Calls the above inline device functions)
//////////////////////////////////////////////////////////////////////////////

template <int ignoreAdetection>
__global__ void MCMLKernel(SimState d_state, GPUThreadStates tstates)
{
  // photon structure stored in registers
  PhotonStructGPU photon; 

  // random number seeds
  UINT64 rnd_x;
  UINT32 rnd_a;

  // Flag to indicate if this thread is active
  UINT32 is_active;

  // Restore the thread state from global memory.
  RestoreThreadState(&d_state, &tstates, &photon, &rnd_x, &rnd_a, &is_active);

  //////////////////////////////////////////////////////////////////////////

  // Coalesce consecutive weight drops to the same address.
  UINT32 last_w = 0;
  UINT32 last_ir = 0, last_iz = 0, last_addr = 0;

  //////////////////////////////////////////////////////////////////////////

#ifdef CACHE_A_RZ_IN_SMEM
  // Cache the frequently acessed region of A_rz in the shared memory.
  __shared__ ARZ_SMEM_TY A_rz_shared[MAX_IR*MAX_IZ];

  if (ignoreAdetection == 0)
  {
    // Clear the cache.
    for (int i = threadIdx.x; i < MAX_IR*MAX_IZ; i += blockDim.x)
    {
      A_rz_shared[i] = 0;
    }
    __syncthreads();
  }

#ifdef USE_32B_ELEM_FOR_ARZ_SMEM
  // Overflow handling:
  //
  // It is too spacious to keep track of whether or not each element in
  // the shared memory is about to overflow. Therefore, we divide all the
  // elements into NUM_THREADS_PER_BLOCK groups (cyclic distribution). For
  // each group, we use a single flag to keep track of if ANY element in it
  // is about to overflow. This results in the following array.
  //
  // At the end of each simulation step, if the flag for any of the groups
  // is set, the corresponding thread (with id equal to the group index)
  // flushes ALL elements in the group to the global memory.
  //
  // This array is dynamically allocated.
  //
  UINT32 *A_rz_overflow = (UINT32*)MCMLKernel_smem;
  if (ignoreAdetection == 0)
  {
    // Clear the flags.
    A_rz_overflow[threadIdx.x] = 0;
  }
#endif

#endif

  //////////////////////////////////////////////////////////////////////////

  // Get the copy of A_rz (in the global memory) this thread writes to.
  UINT64 *g_A_rz = d_state.A_rz
    + (blockIdx.x % N_A_RZ_COPIES) * (d_simparam.nz * d_simparam.nr);

  //////////////////////////////////////////////////////////////////////////

  for (int iIndex = 0; iIndex < NUM_STEPS; ++iIndex)
  {
    // Only process photon if the thread is active.
    if (is_active)
    {
      //>>>>>>>>> StepSizeInTissue() in MCML
      ComputeStepSize(&photon,&rnd_x, &rnd_a);

      //>>>>>>>>> HitBoundary() in MCML
      photon.hit = HitBoundary(&photon);

      Hop(&photon);

      if (photon.hit)
      {
        FastReflectTransmit(&photon, &d_state, &rnd_x, &rnd_a);
      }
      else
      {
        //>>>>>>>>> Drop() in MCML
        GFLOAT dwa = photon.w * d_layerspecs[photon.layer].mua_muas;
        photon.w -= dwa;

        if (ignoreAdetection == 0)
        {
          // automatic __float2uint_rz
          UINT32 iz = FAST_DIV(photon.z, d_simparam.dz);
          // automatic __float2uint_rz
          UINT32 ir = FAST_DIV(
            SQRT(photon.x * photon.x + photon.y * photon.y),
            d_simparam.dr);

          // Only record if photon is not at the edge!!
          // This will be ignored anyways.
          if (iz < d_simparam.nz && ir < d_simparam.nr)
          {
            UINT32 addr = ir * d_simparam.nz + iz;

            if (addr != last_addr)
            {
#ifdef CACHE_A_RZ_IN_SMEM
              // Commit the weight drop to memory.
              if (last_ir < MAX_IR && last_iz < MAX_IZ)
              {
                // Write it to the shared memory.
                last_addr = last_ir * MAX_IZ + last_iz;
#ifdef USE_32B_ELEM_FOR_ARZ_SMEM
                // Use 32-bit atomicAdd.
                UINT32 oldval = atomicAdd(&A_rz_shared[last_addr], last_w);
                // Detect overflow.
                if (oldval >= d_simparam.A_rz_overflow)
                {
                  A_rz_overflow[last_addr % blockDim.x] = 1;
                }
#else
                // 64-bit atomic instruction
                AtomicAddULL_Shared(&A_rz_shared[last_addr], last_w);
#endif
              }
              else
#endif
              {
                // Write it to the global memory directly.
                AtomicAddULL_Global(&g_A_rz[last_addr], last_w);
              }

              last_ir = ir; last_iz = iz;
              last_addr = addr;

              // Reset the last weight.
              last_w = 0;
            }

            // Accumulate to the last weight.
            last_w += (UINT32)(dwa * WEIGHT_SCALE);
          }
        }
        //>>>>>>>>> end of Drop()

        Spin(d_layerspecs[photon.layer].g, &photon, &rnd_x, &rnd_a);
      }

      /***********************************************************
      *  >>>>>>>>> Roulette()
      *  If the photon weight is small, the photon packet tries
      *  to survive a roulette.
      ****/
      if (photon.w < WEIGHT)
      {
        GFLOAT rand = rand_MWC_co(&rnd_x, &rnd_a);

        // This photon survives the roulette.
        if (photon.w != MCML_FP_ZERO && rand < CHANCE)
          photon.w *= (FP_ONE / CHANCE);
        // This photon is terminated.
        else if (atomicSub(d_state.n_photons_left, 1) > gridDim.x*blockDim.x)
          LaunchPhoton(&photon); // Launch a new photon.
        // No need to process any more photons.
        else
          is_active = 0;
      }
    }

    //////////////////////////////////////////////////////////////////////////

#if defined(CACHE_A_RZ_IN_SMEM) && defined(USE_32B_ELEM_FOR_ARZ_SMEM)
    if (ignoreAdetection == 0)
    {
      // Enter a phase of handling overflow in A_rz_shared.
      __syncthreads();

      if (A_rz_overflow[threadIdx.x])
      {
        // Flush all elements I am responsible for to the global memory.
        for (int i = threadIdx.x; i < MAX_IR*MAX_IZ; i += blockDim.x)
        {
          Flush_Arz(g_A_rz, A_rz_shared, i);
          A_rz_shared[i] = 0;
        }
        // Reset the flag.
        A_rz_overflow[threadIdx.x] = 0;
      }

      __syncthreads();
    }
#endif

    //////////////////////////////////////////////////////////////////////
  } // end of the main loop

  __syncthreads();

  if (ignoreAdetection == 0)
  {
    // Commit the last weight drop.
    // NOTE: last_w == 0 if inactive.
    if (last_w > 0)
    {
      // Commit to the global memory directly.
      // TODO: could we commit it to the shared memory, or does it matter?
      AtomicAddULL_Global(&g_A_rz[last_addr], last_w);
    }
  }

  //////////////////////////////////////////////////////////////////////////

#ifdef CACHE_A_RZ_IN_SMEM
  if (ignoreAdetection == 0)
  {
    // Flush A_rz_shared to the global memory.
    for (int i = threadIdx.x; i < MAX_IR*MAX_IZ; i += blockDim.x)
    {
      Flush_Arz(g_A_rz, A_rz_shared, i);
    }
  }
#endif

  //////////////////////////////////////////////////////////////////////////

  // Save the thread state to the global memory.
  SaveThreadState(&d_state, &tstates, &photon, rnd_x, is_active);
}

//////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////

__global__ void sum_A_rz(UINT64 *g_A_rz)
{
  UINT64 sum;

  int n_elems = d_simparam.nz * d_simparam.nr;
  int base_ofst, ofst;

  for (base_ofst = blockIdx.x * blockDim.x + threadIdx.x;
      base_ofst < n_elems; base_ofst += blockDim.x * gridDim.x)
  {
    sum = 0;
    ofst = base_ofst;
#pragma unroll
    for (int i = 0; i < N_A_RZ_COPIES; ++i)
    {
      sum += g_A_rz[ofst];
      ofst += n_elems;
    }
    g_A_rz[base_ofst] = sum;
  }
}

#endif  // _GPUMCML_KERNEL_CU_

