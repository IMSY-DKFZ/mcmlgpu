#include "hip/hip_runtime.h"
/*****************************************************************************
*
*   GPU memory allocation, initialization, and transfer (Host <--> GPU)
*
****************************************************************************/
/*	 
*   This file is part of GPUMCML.
* 
*   GPUMCML is free software: you can redistribute it and/or modify
*   it under the terms of the GNU General Public License as published by
*   the Free Software Foundation, either version 3 of the License, or
*   (at your option) any later version.
*
*   GPUMCML is distributed in the hope that it will be useful,
*   but WITHOUT ANY WARRANTY; without even the implied warranty of
*   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
*   GNU General Public License for more details.
*
*   You should have received a copy of the GNU General Public License
*   along with GPUMCML.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <cstdio>

#include "gpumcml_kernel.h"

//////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////
//   Initialize Device Constant Memory with read-only data
//////////////////////////////////////////////////////////////////////////////
int InitDCMem(SimulationStruct *sim, UINT32 A_rz_overflow) {
    // Make sure that the number of layers is within the limit.
    UINT32 n_layers = sim->n_layers + 2;
    if (n_layers > MAX_LAYERS) return 1;

    SimParamGPU h_simparam;

    h_simparam.num_layers = sim->n_layers;  // not plus 2 here
    h_simparam.init_photon_w = sim->start_weight;
    h_simparam.dz = (GFLOAT) sim->det.dz;
    h_simparam.dr = (GFLOAT) sim->det.dr;
    h_simparam.na = sim->det.na;
    h_simparam.nz = sim->det.nz;
    h_simparam.nr = sim->det.nr;
    h_simparam.A_rz_overflow = A_rz_overflow;

    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_simparam),
                                      &h_simparam, sizeof(SimParamGPU)));

    LayerStructGPU h_layerspecs[MAX_LAYERS];

    for (UINT32 i = 0; i < n_layers; ++i) {
        h_layerspecs[i].z0 = (GFLOAT) sim->layers[i].z_min;
        h_layerspecs[i].z1 = (GFLOAT) sim->layers[i].z_max;
        GFLOAT n1 = (GFLOAT) sim->layers[i].n;
        h_layerspecs[i].n = n1;

        // TODO: sim->layer should not do any pre-computation.
        GFLOAT rmuas = (GFLOAT) sim->layers[i].mutr;
        h_layerspecs[i].muas = FP_ONE / rmuas;
        h_layerspecs[i].rmuas = rmuas;
        h_layerspecs[i].mua_muas = (GFLOAT) sim->layers[i].mua * rmuas;

        h_layerspecs[i].g = (GFLOAT) sim->layers[i].g;

        if (i == 0 || i == n_layers - 1) {
            h_layerspecs[i].cos_crit0 = MCML_FP_ZERO;
            h_layerspecs[i].cos_crit1 = MCML_FP_ZERO;
        } else {
            GFLOAT n2 = (GFLOAT) sim->layers[i - 1].n;
            h_layerspecs[i].cos_crit0 = (n1 > n2) ?
                                        sqrtf(FP_ONE - n2 * n2 / (n1 * n1)) : MCML_FP_ZERO;
            n2 = (GFLOAT) sim->layers[i + 1].n;
            h_layerspecs[i].cos_crit1 = (n1 > n2) ?
                                        sqrtf(FP_ONE - n2 * n2 / (n1 * n1)) : MCML_FP_ZERO;
        }
    }

    // Copy layer data to constant device memory
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_layerspecs),
                                      &h_layerspecs, n_layers * sizeof(LayerStructGPU)));

    return 0;
}

//////////////////////////////////////////////////////////////////////////////
//   Initialize Device Memory (global) for read/write data
//////////////////////////////////////////////////////////////////////////////
int InitSimStates(SimState *HostMem, SimState *DeviceMem,
                  GPUThreadStates *tstates, SimulationStruct *sim,
                  int n_threads) {
    int rz_size = sim->det.nr * sim->det.nz;
    int ra_size = sim->det.nr * sim->det.na;

    unsigned int size;

    // Allocate n_photons_left (on device only)
    size = sizeof(UINT32);
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    CUDA_SAFE_CALL(hipMalloc((void **) &DeviceMem->n_photons_left, size));
    CUDA_SAFE_CALL(hipMemcpy(DeviceMem->n_photons_left,
                              HostMem->n_photons_left, size, hipMemcpyHostToDevice));

    // random number generation (on device only)
    size = n_threads * sizeof(UINT32);
    CUDA_SAFE_CALL(hipMalloc((void **) &DeviceMem->a, size));
    CUDA_SAFE_CALL(hipMemcpy(DeviceMem->a, HostMem->a, size,
                              hipMemcpyHostToDevice));
    size = n_threads * sizeof(UINT64);
    CUDA_SAFE_CALL(hipMalloc((void **) &DeviceMem->x, size));
    CUDA_SAFE_CALL(hipMemcpy(DeviceMem->x, HostMem->x, size,
                              hipMemcpyHostToDevice));


    // Allocate A_rz on host and device
    size = rz_size * sizeof(UINT64);
    HostMem->A_rz = (UINT64 *) malloc(size);
    if (HostMem->A_rz == NULL) {
        fprintf(stderr, "Error allocating HostMem->A_rz");
        exit(1);
    }
    // On the device, we allocate multiple copies for less access contention.
    size *= N_A_RZ_COPIES;
    CUDA_SAFE_CALL(hipMalloc((void **) &DeviceMem->A_rz, size));
    CUDA_SAFE_CALL(hipMemset(DeviceMem->A_rz, 0, size));

    // Allocate Rd_ra on host and device
    size = ra_size * sizeof(UINT64);
    HostMem->Rd_ra = (UINT64 *) malloc(size);
    if (HostMem->Rd_ra == NULL) {
        printf("Error allocating HostMem->Rd_ra");
        exit(1);
    }
    CUDA_SAFE_CALL(hipMalloc((void **) &DeviceMem->Rd_ra, size));
    CUDA_SAFE_CALL(hipMemset(DeviceMem->Rd_ra, 0, size));

    // Allocate Tt_ra on host and device
    size = ra_size * sizeof(UINT64);
    HostMem->Tt_ra = (UINT64 *) malloc(size);
    if (HostMem->Tt_ra == NULL) {
        printf("Error allocating HostMem->Tt_ra");
        exit(1);
    }
    CUDA_SAFE_CALL(hipMalloc((void **) &DeviceMem->Tt_ra, size));
    CUDA_SAFE_CALL(hipMemset(DeviceMem->Tt_ra, 0, size));

    /* Allocate and initialize GPU thread states on the device.
    *
    * We only initialize rnd_a and rnd_x here. For all other fields, whose
    * initial value is a known constant, we use a kernel to do the
    * initialization.
    */

    // photon structure
    size = n_threads * sizeof(GFLOAT);
    CUDA_SAFE_CALL(hipMalloc((void **) &tstates->photon_x, size));
    CUDA_SAFE_CALL(hipMalloc((void **) &tstates->photon_y, size));
    CUDA_SAFE_CALL(hipMalloc((void **) &tstates->photon_z, size));
    CUDA_SAFE_CALL(hipMalloc((void **) &tstates->photon_ux, size));
    CUDA_SAFE_CALL(hipMalloc((void **) &tstates->photon_uy, size));
    CUDA_SAFE_CALL(hipMalloc((void **) &tstates->photon_uz, size));
    CUDA_SAFE_CALL(hipMalloc((void **) &tstates->photon_w, size));
    size = n_threads * sizeof(UINT32);
    CUDA_SAFE_CALL(hipMalloc((void **) &tstates->photon_layer, size));

    // thread active
    CUDA_SAFE_CALL(hipMalloc((void **) &tstates->is_active, size));

    return 1;
}

//////////////////////////////////////////////////////////////////////////////
//   Transfer data from Device to Host memory after simulation
//////////////////////////////////////////////////////////////////////////////
int CopyDeviceToHostMem(SimState *HostMem, SimState *DeviceMem,
                        SimulationStruct *sim, int n_threads) {
    int rz_size = sim->det.nr * sim->det.nz;
    int ra_size = sim->det.nr * sim->det.na;

    // Copy A_rz, Rd_ra and Tt_ra
    CUDA_SAFE_CALL(hipMemcpy(HostMem->A_rz, DeviceMem->A_rz, rz_size * sizeof(UINT64), hipMemcpyDeviceToHost));
    CUDA_SAFE_CALL(hipMemcpy(HostMem->Rd_ra, DeviceMem->Rd_ra, ra_size * sizeof(UINT64), hipMemcpyDeviceToHost));
    CUDA_SAFE_CALL(hipMemcpy(HostMem->Tt_ra, DeviceMem->Tt_ra, ra_size * sizeof(UINT64), hipMemcpyDeviceToHost));

    //Also copy the state of the RNG's
    CUDA_SAFE_CALL(
            hipMemcpy(HostMem->x, DeviceMem->x, n_threads * sizeof(UINT64),
                       hipMemcpyDeviceToHost));

    return 0;
}

//////////////////////////////////////////////////////////////////////////////
//   Free Host Memory
//////////////////////////////////////////////////////////////////////////////
void FreeHostSimState(SimState *hstate) {
    if (hstate->n_photons_left != NULL) {
        free(hstate->n_photons_left);
        hstate->n_photons_left = NULL;
    }

    // DO NOT FREE RANDOM NUMBER SEEDS HERE.

    if (hstate->A_rz != NULL) {
        free(hstate->A_rz);
        hstate->A_rz = NULL;
    }
    if (hstate->Rd_ra != NULL) {
        free(hstate->Rd_ra);
        hstate->Rd_ra = NULL;
    }
    if (hstate->Tt_ra != NULL) {
        free(hstate->Tt_ra);
        hstate->Tt_ra = NULL;
    }
}

//////////////////////////////////////////////////////////////////////////////
//   Free GPU Memory
//////////////////////////////////////////////////////////////////////////////
void FreeDeviceSimStates(SimState *dstate, GPUThreadStates *tstates) {
    hipFree(dstate->n_photons_left);
    dstate->n_photons_left = NULL;

    hipFree(dstate->x);
    dstate->x = NULL;
    hipFree(dstate->a);
    dstate->a = NULL;

    hipFree(dstate->A_rz);
    dstate->A_rz = NULL;
    hipFree(dstate->Rd_ra);
    dstate->Rd_ra = NULL;
    hipFree(dstate->Tt_ra);
    dstate->Tt_ra = NULL;

    hipFree(tstates->photon_x);
    tstates->photon_x = NULL;
    hipFree(tstates->photon_y);
    tstates->photon_y = NULL;
    hipFree(tstates->photon_z);
    tstates->photon_z = NULL;
    hipFree(tstates->photon_ux);
    tstates->photon_ux = NULL;
    hipFree(tstates->photon_uy);
    tstates->photon_uy = NULL;
    hipFree(tstates->photon_uz);
    tstates->photon_uz = NULL;
    hipFree(tstates->photon_w);
    tstates->photon_w = NULL;
    hipFree(tstates->photon_layer);
    tstates->photon_layer = NULL;
    hipFree(tstates->is_active);
    tstates->is_active = NULL;
}

//////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////

